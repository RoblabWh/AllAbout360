#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "mapping.h"

#ifdef DEBUG
hipError_t _cuda_error;
#define CUDA_ERROR_CHECK(FNC_CALL) _cuda_error = FNC_CALL; if(_cuda_error != 0) { printf("%s: %s\n",  #FNC_CALL, hipGetErrorString(_cuda_error)); exit(EXIT_FAILURE); }
#define CUDA_ERROR_CHECK_KERNEL(KERNEL_CALL...) CUDA_ERROR_CHECK_KERNEL_( (KERNEL_CALL) )
#define CUDA_ERROR_CHECK_KERNEL_(KERNEL_CALL) KERNEL_CALL; _cuda_error = hipGetLastError(); if(_cuda_error != 0) { printf("%s: %s\n", #KERNEL_CALL, hipGetErrorString(_cuda_error)); exit(EXIT_FAILURE); }
#else
#define CUDA_ERROR_CHECK(FNC_CALL) FNC_CALL
#define CUDA_ERROR_CHECK_KERNEL(KERNEL_CALL...) KERNEL_CALL
#endif

const unsigned int THREADS_PER_BLOCK = 1024;

double *d_map;
unsigned char *d_idata, *d_odata;
unsigned short h_width, h_height;
unsigned int h_elelen, h_memlen, h_memlen_2;
unsigned short num_blocks;

template<interpolation_type interpol_t>
__global__ void d_remap(unsigned char *g_idata_1, unsigned char *g_idata_2, unsigned char *g_odata, double *g_map, unsigned int len)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < len) // fitting number of blocks / threads
    {
        unsigned int i3 = i * 3;
        unsigned int i14 = i * 14;
        unsigned int idx1 = g_map[i14];
        unsigned int idx2 = g_map[i14 + 2];
        double bf1 = g_map[i14 + 4]; // float maybe
        double bf2 = g_map[i14 + 5];
        if (interpol_t == interpolation_type::NEAREST_NEIGHBOUR)
        {
            g_odata[i3] = g_idata_1[idx1] * bf1 + g_idata_2[idx2] * bf2;
            g_odata[i3 + 1] = g_idata_1[idx1 + 1] * bf1 + g_idata_2[idx2 + 1] * bf2;
            g_odata[i3 + 2] = g_idata_1[idx1 + 2] * bf1 + g_idata_2[idx2 + 2] * bf2;
        }
        else if (interpol_t == interpolation_type::BILINEAR)
        {
            unsigned int idx1_y1 = g_map[i14 + 1];
            unsigned int idx2_y1 = g_map[i14 + 3];
            unsigned int idx1_x1 = idx1 + 3;
            unsigned int idx2_x1 = idx2 + 3;
            unsigned int idx1_x1y1 = idx1_y1 + 3;
            unsigned int idx2_x1y1 = idx2_y1 + 3;
            double f11 = g_map[i14 + 6], f12 = g_map[i14 + 7],f13 = g_map[i14 + 8],f14 = g_map[i14 + 9];
            double f21 = g_map[i14 + 10], f22 = g_map[i14 + 11],f23 = g_map[i14 + 12],f24 = g_map[i14 + 13];

            g_odata[i3] = (g_idata_1[idx1] * f11 + g_idata_1[idx1_x1] * f12 + g_idata_1[idx1_y1] * f13 + g_idata_1[idx1_x1y1] * f14) * bf1
                        + (g_idata_2[idx2] * f21 + g_idata_2[idx2_x1] * f22 + g_idata_2[idx2_y1] * f23 + g_idata_2[idx2_x1y1] * f24) * bf2;
            g_odata[i3 + 1] = (g_idata_1[idx1 + 1] * f11 + g_idata_1[idx1_x1 + 1] * f12 + g_idata_1[idx1_y1 + 1] * f13 + g_idata_1[idx1_x1y1 + 1] * f14) * bf1
                            + (g_idata_2[idx2 + 1] * f21 + g_idata_2[idx2_x1 + 1] * f22 + g_idata_2[idx2_y1 + 1] * f23 + g_idata_2[idx2_x1y1 + 1] * f24) * bf2;
            g_odata[i3 + 2] = (g_idata_1[idx1 + 2] * f11 + g_idata_1[idx1_x1 + 2] * f12 + g_idata_1[idx1_y1 + 2] * f13 + g_idata_1[idx1_x1y1 + 2] * f14) * bf1
                            + (g_idata_2[idx2 + 2] * f21 + g_idata_2[idx2_x1 + 2] * f22 + g_idata_2[idx2_y1 + 2] * f23 + g_idata_2[idx2_x1y1 + 2] * f24) * bf2;
        }
    }
}

void init_device_memory(const void *map, unsigned short width, unsigned short height)
{
    h_elelen = width * height;
    h_memlen = h_elelen * 3;
    h_memlen_2 = h_memlen / 2;
    num_blocks = h_elelen % THREADS_PER_BLOCK == 0 ? h_elelen / THREADS_PER_BLOCK : h_elelen / THREADS_PER_BLOCK + 1;
    unsigned int maplen = h_elelen * sizeof(double) * 14;

    DBG( printf("width:%u height:%u elements:%u memory:%u blocks:%u maplen:%u \n", width, height, h_elelen, h_memlen, num_blocks, maplen) );

    CUDA_ERROR_CHECK( hipMalloc((void **)&d_idata, h_memlen) );
    CUDA_ERROR_CHECK( hipMalloc((void **)&d_odata, h_memlen) );

    CUDA_ERROR_CHECK( hipMalloc((void **)&d_map, maplen) );
    CUDA_ERROR_CHECK( hipMemcpy(d_map, map, maplen, hipMemcpyHostToDevice) );
}

void free_device_memory()
{
    CUDA_ERROR_CHECK( hipFree(d_idata) );
    CUDA_ERROR_CHECK( hipFree(d_odata) );

    CUDA_ERROR_CHECK( hipFree(d_map) );
}

/**
 * @brief Remaps the given image based on the given mappingfile.
 * 
 * @tparam interpol_t interpolation type
 * @param in Dualfisheye image
 * @param out Equirectangular image
 */
template <interpolation_type interpol_t>
void cuda_remap(const unsigned char *in, unsigned char *out)
{
    CUDA_ERROR_CHECK( hipMemcpy(d_idata, in, h_memlen, hipMemcpyHostToDevice) );

    CUDA_ERROR_CHECK_KERNEL( d_remap<interpol_t><<<num_blocks, THREADS_PER_BLOCK>>>(d_idata, d_idata, d_odata, d_map, h_elelen) );

    CUDA_ERROR_CHECK( hipMemcpy(out, d_odata, h_memlen, hipMemcpyDeviceToHost) );
}

/**
 * @brief Remaps the given images based on the given mappingfile.
 * 
 * @tparam interpol_t interpolation type
 * @param in_1 first fisheye image
 * @param in_2 second fisheye image
 * @param out Equirectangular image
 */
template <interpolation_type interpol_t>
void cuda_remap(const unsigned char *in_1, const unsigned char *in_2, unsigned char *out)
{
    CUDA_ERROR_CHECK( hipMemcpy(d_idata, in_1, h_memlen_2, hipMemcpyHostToDevice) );
    CUDA_ERROR_CHECK( hipMemcpy(d_idata + h_memlen_2, in_2, h_memlen_2, hipMemcpyHostToDevice) );

    CUDA_ERROR_CHECK_KERNEL( d_remap<interpol_t><<<num_blocks, THREADS_PER_BLOCK>>>(d_idata, d_idata + h_memlen_2, d_odata, d_map, h_elelen) );

    CUDA_ERROR_CHECK( hipMemcpy(out, d_odata, h_memlen, hipMemcpyDeviceToHost) );
}

void cuda_remap_nn(const unsigned char *in, unsigned char *out) { cuda_remap<interpolation_type::NEAREST_NEIGHBOUR>(in, out); }
void cuda_remap_li(const unsigned char *in, unsigned char *out) { cuda_remap<interpolation_type::BILINEAR>(in, out); }
void cuda_remap_nn(const unsigned char *in_1, const unsigned char *in_2, unsigned char *out) { cuda_remap<interpolation_type::NEAREST_NEIGHBOUR>(in_1, in_2, out); }
void cuda_remap_li(const unsigned char *in_1, const unsigned char *in_2, unsigned char *out) { cuda_remap<interpolation_type::BILINEAR>(in_1, in_2, out); }