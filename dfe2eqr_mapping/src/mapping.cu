#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "mapping.h"

#ifdef DEBUG
#define DBG(E...) E
hipError_t _cuda_error;
#define CUDA_ERROR_CHECK(FNC_CALL) _cuda_error = FNC_CALL; if(_cuda_error != 0) { printf("%s: %s\n",  #FNC_CALL, hipGetErrorString(_cuda_error)); exit(EXIT_FAILURE); }
#define CUDA_ERROR_CHECK_KERNEL(KERNEL_CALL...) CUDA_ERROR_CHECK_KERNEL_( (KERNEL_CALL) )
#define CUDA_ERROR_CHECK_KERNEL_(KERNEL_CALL) KERNEL_CALL; _cuda_error = hipGetLastError(); if(_cuda_error != 0) { printf("%s: %s\n", #KERNEL_CALL, hipGetErrorString(_cuda_error)); exit(EXIT_FAILURE); }
#else
#define DBG(E...)
#define CUDA_ERROR_CHECK(FNC_CALL) FNC_CALL
#define CUDA_ERROR_CHECK_KERNEL(KERNEL_CALL...) KERNEL_CALL
#endif

const unsigned int THREADS_PER_BLOCK = 1024;

unsigned short *d_mapx, *d_mapy;
unsigned char *d_idata, *d_odata;
unsigned short h_width, h_height;
unsigned int h_elelen, h_memlen, h_memlen_2;
unsigned short num_blocks;

__global__
void d_map(unsigned char *g_idata, unsigned char *g_odata, unsigned short *g_mapx, unsigned short *g_mapy, unsigned int size_row, unsigned int len)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < len)
    {
        unsigned int index = (g_mapy[i] * size_row + g_mapx[i]) * 3;
        unsigned int i3 = i * 3;
        g_odata[i3] = g_idata[index];
        g_odata[i3 + 1] = g_idata[index + 1];
        g_odata[i3 + 2] = g_idata[index + 2];
    }
}

__global__
void d_map(unsigned char *g_idataf, unsigned char *g_idatar, unsigned char *g_odata, unsigned short *g_mapx, unsigned short *g_mapy, unsigned int size_row, unsigned int len)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < len)
	{
		unsigned int index;
		unsigned int i3 = i * 3;
		if (g_mapx[i] < size_row)
		{
			index = (g_mapy[i] * size_row + g_mapx[i]) * 3;
			g_odata[i3] = g_idataf[index];
			g_odata[i3 + 1] = g_idataf[index + 1];
			g_odata[i3 + 2] = g_idataf[index + 2];
		}
		else
		{
			index = (g_mapy[i] * size_row + g_mapx[i] - size_row) * 3;
			g_odata[i3] = g_idatar[index];
			g_odata[i3 + 1] = g_idatar[index + 1];
			g_odata[i3 + 2] = g_idatar[index + 2];
		}
	}
}

void init_device_memory(const std::vector<unsigned short>& mapx, const std::vector<unsigned short>& mapy, 
    unsigned short width, unsigned short height)
{
    init_device_memory(mapx.data(), mapy.data(), width, height);
}

void init_device_memory(const unsigned short *mapx, const unsigned short *mapy, unsigned short width, unsigned short height)
{
    h_width = width;
    h_height = height;
    h_elelen = width * height;
    h_memlen = h_elelen * 3;
    h_memlen_2 = h_memlen / 2;
    num_blocks = h_elelen % THREADS_PER_BLOCK == 0 ? h_elelen / THREADS_PER_BLOCK : h_elelen / THREADS_PER_BLOCK + 1;
    unsigned int maplen = h_elelen * sizeof(unsigned short);

    DBG( printf("width:%u height:%u elements:%u memory:%u blocks:%u \n", h_width, h_height, h_elelen, h_memlen, num_blocks) );

    CUDA_ERROR_CHECK( hipMalloc((void **)&d_idata, h_memlen) );
    CUDA_ERROR_CHECK( hipMalloc((void **)&d_odata, h_memlen) );

    CUDA_ERROR_CHECK( hipMalloc((void **)&d_mapx, maplen) );
    CUDA_ERROR_CHECK( hipMalloc((void **)&d_mapy, maplen) );

    CUDA_ERROR_CHECK( hipMemcpy(d_mapx, mapx, maplen, hipMemcpyHostToDevice) );
    CUDA_ERROR_CHECK( hipMemcpy(d_mapy, mapy, maplen, hipMemcpyHostToDevice) );
}

void free_device_memory()
{
    CUDA_ERROR_CHECK( hipFree(d_idata) );
    CUDA_ERROR_CHECK( hipFree(d_odata) );

    CUDA_ERROR_CHECK( hipFree(d_mapx) );
    CUDA_ERROR_CHECK( hipFree(d_mapy) );
}


void map(const unsigned char *in_img, unsigned char *out_img)
{
    CUDA_ERROR_CHECK( hipMemcpy(d_idata, in_img, h_memlen, hipMemcpyHostToDevice) );
    
    CUDA_ERROR_CHECK_KERNEL( d_map<<<num_blocks, THREADS_PER_BLOCK>>>(d_idata, d_odata, d_mapx, d_mapy, h_width, h_elelen) );

    CUDA_ERROR_CHECK( hipMemcpy(out_img, d_odata, h_memlen, hipMemcpyDeviceToHost) );
}

void map(const unsigned char *in_img_front, const unsigned char *in_img_rear, unsigned char *out_img)
{
    CUDA_ERROR_CHECK( hipMemcpy(d_idata, in_img_front, h_memlen_2, hipMemcpyHostToDevice) );
    CUDA_ERROR_CHECK( hipMemcpy(d_idata + h_memlen_2, in_img_rear, h_memlen_2, hipMemcpyHostToDevice) );

    CUDA_ERROR_CHECK_KERNEL( d_map<<<num_blocks, THREADS_PER_BLOCK>>>(d_idata, d_idata + h_memlen_2, d_odata, d_mapx, d_mapy, h_height, h_elelen) );

    CUDA_ERROR_CHECK( hipMemcpy(out_img, d_odata, h_memlen, hipMemcpyDeviceToHost) );
}